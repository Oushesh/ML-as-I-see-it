#include "hip/hip_runtime.h"
//gpu_multiply.cu

#include <iostream>

#define N 500000000
#define THREDS_PER_BLCOCK 1024

/*
GPU Kernel Function to multiply 2
array elements and also update the results
on the second array
*/

//This is usually called the kernel
__global__ void multiply()
{
  int index = threadIdx.x + blockIdx.x*blockDim.x;
  if (index<n)
  {
    q[index]=p[index]*q[index];
  }
}

int main()
{
  double *p,*q; //host copies of p,q
  double *gpu_p,*gpu_q;//device copies or gpu copies of p,q

  //hipMalloc
  hipMalloc((void**)&gpu_p,size);
  hipMalloc((void**)&gpu_q,size);


  //Allocate CPU/host copies of p,q
  p = (double *)malloc(size);
  q = (double *)malloc(size);

  //After allocating memory for the CPU/host
  //copies, let's now assign 2 values for both
  //arrays. The "cudaMempyHostToDevice" function
  //transfers host data to device:


  //Setup input values

  for (int i=0;i<N;i++)
  {
    p[i]=24.0;
    q[i]=12.0;

    //Copy inputs to device
    
  }
}
